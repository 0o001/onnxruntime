#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/tensor/mode_restore_impl.h"
#include "core/providers/cuda/cu_inc/bitmask.cuh"
#include <hipcub/hipcub.hpp>

namespace onnxruntime {
namespace cuda {

void GetZeroPointRestoreTempStorageBytesImpl(hipStream_t stream,
                                             size_t& temp_storage_bytes,
                                             int total_element_count) {
  hipcub::DeviceScan::InclusiveSum(
      static_cast<void*>(nullptr),  // input, when NULL, the required allocation size is written to temp_storage_bytes and no work is done.
      temp_storage_bytes,           // input or output
      static_cast<int*>(nullptr),   // input
      static_cast<int*>(nullptr),   // output
      total_element_count,          // input
      stream);
}

void CalculateInputOffsetForEachOutputImpl(hipStream_t stream,
                                           void* d_temp_storage,
                                           size_t& temp_storage_bytes,
                                           int* restored_output_mask,
                                           int* output_idx_to_input_idx_map_buffer,
                                           int total_element_count) {
  hipcub::DeviceScan::InclusiveSum(
      d_temp_storage,                      // input, when NULL, the required allocation size is written to temp_storage_bytes and no work is done.
      temp_storage_bytes,                  // input or output
      restored_output_mask,                // input
      output_idx_to_input_idx_map_buffer,  // output
      total_element_count,                 // input
      stream);
}

constexpr int kBlockSize = 256;
constexpr int kNumUnroll = 4;

__global__ void FillOutputFromMaskKernel(const CUDA_LONG N,
                                         const fast_divmod fdm_bits_per_element,
                                         const BitmaskElementType* mask_data,
                                         int* restored_output_mask) {
  CUDA_LONG idx = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_LONG id = idx * kNumUnroll;

  int masks[kNumUnroll];
  if (id < N) {
    int bitmask_idx, bitmask_shift;
    fdm_bits_per_element.divmod(id, bitmask_idx, bitmask_shift);
    BitmaskElementType shifted_mask = mask_data[bitmask_idx] >> bitmask_shift;
#pragma unroll
    for (int i = 0; i < kNumUnroll; i++) {
      masks[i] = ((shifted_mask & (1 << i)) != 0);
    }
  }

#pragma unroll
  for (int i = 0; i < kNumUnroll; ++i) {
    CUDA_LONG li = id + i;
    if (li < N) {
      restored_output_mask[li] = masks[i];
      // printf("restored_output_mask[%d] = %d \n", static_cast<int>(li), restored_output_mask[li]);
    }
  }
}

void FillOutputFromMaskImpl(hipStream_t stream,
                            const int64_t total_element_count,
                            const BitmaskElementType* mask_data,
                            int* restored_output_mask) {
  const int blocksPerGrid = static_cast<int>(CeilDiv(total_element_count, kBlockSize * kNumUnroll));
  fast_divmod fdm_bits_per_element(kNumBitsPerBitmaskElement);
  FillOutputFromMaskKernel<<<blocksPerGrid, kBlockSize, 0, stream>>>(
      static_cast<CUDA_LONG>(total_element_count), fdm_bits_per_element, mask_data, restored_output_mask);
}

template <typename T>
__global__ void RestoreFromMaskKernel(const CUDA_LONG N,
                                      const int step_size,
                                      const int steps_per_thread,
                                      const fast_divmod fdm_bits_per_element,
                                      const float zero_point_value,
                                      const T* input_data,
                                      const int* output_idx_to_input_idx_map_buffer,
                                      T* output_data) {
  CUDA_LONG idx = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = 0; i < steps_per_thread; ++i) {
    CUDA_LONG id = idx * kNumUnroll + i * step_size;

    int maps[kNumUnroll + 1];
    if (id < N) {
      if (id == 0) {
        maps[0] = 0;
      } else {
        maps[0] = output_idx_to_input_idx_map_buffer[id - 1];
      }

#pragma unroll
      for (int i = 0; i < kNumUnroll; ++i) {
        CUDA_LONG li = id + i;
        if (li < N) {
          maps[i + 1] = output_idx_to_input_idx_map_buffer[li];
        }
      }
    }

#pragma unroll
    for (int i = 0; i < kNumUnroll; ++i) {
      CUDA_LONG li = id + i;
      if (li < N) {
        int map_value = maps[i + 1] - maps[i];
        output_data[li] = map_value == 1 ? input_data[maps[i]] : static_cast<T>(zero_point_value);
        // printf("output_data[%d] = %f, map_value: %d, maps[i + 1]: %d \n", static_cast<int>(li), static_cast<float>(output_data[li]), map_value, maps[i + 1]);
      }
    }
  }
}

template <typename T>
void RestoreFromMaskImpl(const hipDeviceProp_t& prop,
                         hipStream_t stream,
                         const int64_t total_element_count,
                         const float zero_point_value,
                         const T* input_data,
                         const int* output_idx_to_input_idx_map_buffer,
                         T* output_data) {
  const int blocks_per_sm = prop.maxThreadsPerMultiProcessor / kBlockSize;
  const int grid_size =
      std::min(prop.multiProcessorCount * blocks_per_sm,
               static_cast<int>(CeilDiv(total_element_count, kBlockSize * kNumUnroll)));

  const int step_size = kBlockSize * grid_size * kNumUnroll;
  const int steps_per_thread = static_cast<int>(CeilDiv(total_element_count, step_size));
  fast_divmod fdm_bits_per_element(kNumBitsPerBitmaskElement);

  // std::cout << "step_size: " << step_size << ", steps_per_thread: " << steps_per_thread << ", total_element_count: " << total_element_count << std::endl;
  RestoreFromMaskKernel<T><<<grid_size, kBlockSize, 0, stream>>>(
      static_cast<CUDA_LONG>(total_element_count),
      step_size,
      steps_per_thread,
      fdm_bits_per_element,
      zero_point_value,
      input_data,
      output_idx_to_input_idx_map_buffer,
      output_data);
}

#define SPECIALIZED_RESTORE_FROM_MASK_IMPL(T)                                         \
  template void RestoreFromMaskImpl<T>(const hipDeviceProp_t& prop,                    \
                                       hipStream_t stream,                           \
                                       const int64_t total_element_count,             \
                                       const float zero_point_value,                  \
                                       const T* input_data,                           \
                                       const int* output_idx_to_input_idx_map_buffer, \
                                       T* output_data);

SPECIALIZED_RESTORE_FROM_MASK_IMPL(float)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(double)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(half)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(BFloat16)

#undef SPECIALIZED_RESTORE_FROM_MASK_IMPL

}  // namespace cuda
}  // namespace onnxruntime
