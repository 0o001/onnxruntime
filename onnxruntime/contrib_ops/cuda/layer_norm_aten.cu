#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include <hip/hip_fp16.h>
#include <thrust/tuple.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/accumulation_type.h"
#include "layer_norm_impl.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace onnxruntime::cuda;

constexpr int kCUDANumThreads = 256;
//constexpr int kColwiseReduceTileSize = 32;

#if defined(__HIPCC__) || defined(__HIPCC__)
#define C10_HOST_DEVICE __host__ __device__
#define C10_DEVICE __device__
#define C10_HOST __host__
#else
#define C10_HOST_DEVICE
#define C10_HOST
#define C10_DEVICE
#endif

#ifdef __HIP_PLATFORM_HCC__
#define C10_WARP_SIZE 64
#else
#define C10_WARP_SIZE 32
#endif

#if defined(__clang__)
  #define __ubsan_ignore_float_divide_by_zero__ __attribute__((no_sanitize("float-divide-by-zero")))
  //#define __ubsan_ignore_undefined__ __attribute__((no_sanitize("undefined")))
  //#define __ubsan_ignore_signed_int_overflow__ __attribute__((no_sanitize("signed-integer-overflow")))
#else
  #define __ubsan_ignore_float_divide_by_zero__
  //#define __ubsan_ignore_undefined__
  //#define __ubsan_ignore_signed_int_overflow__
#endif

#if defined(__HIP_PLATFORM_HCC__)
// take these out when ROCm implements std:: math functions
#include <math.h>
template <typename scalar_t>
static __forceinline__ __device__ scalar_t device_sqrt(scalar_t val);

template <>
__forceinline__ __device__ float device_sqrt(float val) {
  return ::sqrtf(val);
}

template <>
__forceinline__ __device__ double device_sqrt(double val) {
  return ::sqrt(val);
}
#else
template<typename scalar_t>
__forceinline__ __device__ double device_sqrt(scalar_t val) {
  return std::sqrt(val);
}
#endif

#if defined(__HIPCC__) || defined(__HIPCC__)
#include <thrust/pair.h>
#else
#include <cmath>
#define device_sqrt std::sqrt
#endif

/*
template <typename T, bool is_cuda>
struct AccumulateType { };

#if defined(__HIPCC__) || defined(__HIPCC__)
template <> struct AccumulateType<half, true> { using type = float; };
#endif

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
template <> struct AccumulateType<hip_bfloat16, true> {using type = float; };
#endif

template <> struct AccumulateType<float, true> { using type = float; };
template <> struct AccumulateType<double, true> { using type = double; };
template <> struct AccumulateType<int8_t, true> { using type = int64_t; };
template <> struct AccumulateType<uint8_t, true> { using type = int64_t; };
template <> struct AccumulateType<char, true> { using type = int64_t; };
template <> struct AccumulateType<int16_t, true> { using type = int64_t; };
template <> struct AccumulateType<int32_t, true> { using type = int64_t; };
template <> struct AccumulateType<int64_t, true> { using type = int64_t; };
template <> struct AccumulateType<bool, true> {using type = bool; };

template <> struct AccumulateType<float, false> { using type = double; };
template <> struct AccumulateType<double, false> { using type = double; };
template <> struct AccumulateType<int8_t, false> { using type = int64_t; };
template <> struct AccumulateType<uint8_t, false> { using type = int64_t; };
template <> struct AccumulateType<char, false> { using type = int64_t; };
template <> struct AccumulateType<int16_t, false> { using type = int64_t; };
template <> struct AccumulateType<int32_t, false> { using type = int64_t; };
template <> struct AccumulateType<int64_t, false> { using type = int64_t; };
template <> struct AccumulateType<bool, false> {using type = bool; };

template<typename T, bool is_cuda>
using acc_type = typename AccumulateType<T, is_cuda>::type;
*/


constexpr int kCUDABlockReduceNumThreads = 512;

// Sums `val` accross all threads in a warp.
//
// Assumptions:
//   - The size of each block should be a multiple of `C10_WARP_SIZE`
template <typename T>
__inline__ __device__ T WarpReduceSum(T val) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val += WARP_SHFL_DOWN(val, offset);
  }
  return val;
}

// Sums `val` accross all threads in a block.
//
// Assumptions:
//   - Thread blocks are an 1D set of threads (indexed with `threadIdx.x` only)
//   - The size of each block should be a multiple of `C10_WARP_SIZE`
//   - `shared` should be a pointer to shared memory with size of, at least,
//     `sizeof(T) * number_of_warps`
template <typename T>
__inline__ __device__ T BlockReduceSum(T val, T* shared) {
  const int lid = threadIdx.x % C10_WARP_SIZE;
  const int wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduceSum(val);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < blockDim.x / C10_WARP_SIZE) ? shared[lid] : 0;
  if (wid == 0) {
    val = WarpReduceSum(val);
  }
  return val;
}

template <typename T, class ReduceOp>
__inline__ __device__ T WarpReduce(T val, const ReduceOp& op) {
#pragma unroll
  for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
    val = op.combine(val, op.warp_shfl_down(val, offset));
  }
  return val;
}

template <typename T, class ReduceOp>
__inline__ __device__ T
BlockReduce(T val, const ReduceOp& op, const T& identity_element, T* shared) {
  const int lid = threadIdx.x % C10_WARP_SIZE;
  const int wid = threadIdx.x / C10_WARP_SIZE;
  val = WarpReduce(val, op);
  __syncthreads();
  if (lid == 0) {
    shared[wid] = val;
  }
  __syncthreads();
  val = (threadIdx.x < blockDim.x / C10_WARP_SIZE) ? shared[lid]
                                                   : identity_element;
  if (wid == 0) {
    val = WarpReduce(val, op);
  }
  return val;
}


template <typename scalar_t, typename index_t, typename combine_t>
struct WelfordData {
  scalar_t mean;
  scalar_t m2;
  index_t n;
  combine_t nf;

  C10_HOST_DEVICE WelfordData() : mean(0), m2(0), n(0), nf(0) {}

  C10_HOST_DEVICE WelfordData(
      scalar_t mean,
      scalar_t m2,
      index_t n,
      combine_t nf)
      : mean(mean), m2(m2), n(n), nf(nf) {}
};

template <typename scalar_t, typename acc_scalar_t, typename index_t, typename combine_t, typename res_t>
struct WelfordOps {
  index_t correction;
  bool take_sqrt;
 public:
  using acc_t = WelfordData<acc_scalar_t, index_t, combine_t>;
  inline C10_DEVICE acc_t reduce(acc_t acc, scalar_t data, index_t /*idx*/) const {
    acc_scalar_t delta = data - acc.mean;
    // using acc.nf(combine_t) here, as acc.n(index_t) would still be converted
    // accumulation in reduce is done through index_T
    acc_scalar_t new_mean = acc.mean + delta / (acc.nf + 1);
    acc_scalar_t new_delta = data - new_mean;
    return {
      new_mean,
      acc.m2 + delta * new_delta,
      acc.n + 1,
      combine_t(acc.n + 1), // accumulate for combine_t uses index_t
    };
  }
  inline C10_DEVICE acc_t combine(acc_t a, acc_t b) const {
    if (a.nf == 0) {
      return b;
    }
    if (b.nf == 0) {
      return a;
    }
    acc_scalar_t delta = b.mean - a.mean;
    combine_t new_count = a.nf + b.nf;
    acc_scalar_t nb_over_n = b.nf / new_count;
    return {
      a.mean + delta * nb_over_n,
      a.m2 + b.m2 + delta * delta * a.nf * nb_over_n,
      // setting acc.n as -1 since acc.n might not be able to represent the count
      // correctly within its range, setting it to -1 to avoid confusion
      -1,
      new_count
    };
  }

  inline C10_DEVICE res_t project(acc_t acc) const __ubsan_ignore_float_divide_by_zero__ {
    const auto mean = static_cast<scalar_t>(acc.mean);
    const combine_t divisor = acc.nf > correction ? acc.nf - correction : 0;
    const auto var = acc.m2 / divisor;
    res_t results(take_sqrt ? device_sqrt(var) : var, mean);
    return results;
  }

  static C10_DEVICE acc_t translate_idx(acc_t acc, int64_t /*base_idx*/) {
    return acc;
  }

#if defined(__HIPCC__) || defined(__HIPCC__)
  inline __device__ acc_t warp_shfl_down(acc_t acc, int offset) const {
    return {
      WARP_SHFL_DOWN(acc.mean, offset)
      , WARP_SHFL_DOWN(acc.m2, offset)
      , WARP_SHFL_DOWN(acc.n, offset)
      , WARP_SHFL_DOWN(acc.nf, offset)
    };
  }
#endif
  C10_HOST_DEVICE WelfordOps(index_t correction, bool take_sqrt)
      : correction(correction), take_sqrt(take_sqrt) {}
};


// TODO: move to common.cuh, also upate layer_norm.cuh
template <typename T>
__device__ inline T Rsqrt(const T& x);

template <>
__device__ inline float Rsqrt(const float& x) {
  return rsqrtf(x);
}

template <>
__device__ inline double Rsqrt(const double& x) {
  return rsqrt(x);
}

template <>
__device__ inline half Rsqrt(const half& x) {
#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  return hrsqrt(x);
#else
  return half(rsqrtf(float(x)));
#endif
}




template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    double eps, // T eps,
    const T* X,
    T* mean,
    T* rstd) {
  using T_ACC = AccumulationType_t<T>; //acc_type<T, true>;
  using WelfordType = WelfordData<T_ACC, int64_t, T_ACC>;
  using WelfordOp = WelfordOps<T_ACC, T_ACC, int64_t, T_ACC, thrust::pair<T_ACC, T_ACC>>;

  __shared__
      typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::type val_shared[C10_WARP_SIZE];
  WelfordType* val_shared_ptr = reinterpret_cast<WelfordType*>(val_shared);

  const int64_t i = blockIdx.x;
  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    val = welford_op.reduce(val, static_cast<T_ACC>(X[index]), index);
  }
  val = BlockReduce(
      val,
      welford_op,
      /*identity_element=*/WelfordType(0, 0, 0, 0),
      val_shared_ptr);

  if (threadIdx.x == 0) {
    T_ACC m1;
    T_ACC m2;
    thrust::tie(m2, m1) = welford_op.project(val);
    mean[i] = m1;
    rstd[i] = Rsqrt(m2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const T* beta,
    T* Y) {
  using T_ACC =  AccumulationType_t<T>; //acc_type<T, true>;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
            static_cast<T_ACC>(rstd[i]) * gamma_v + beta_v;
  }
}


template <typename T>
void LaunchLayerNorm(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    T* output,
    T* mean,
    T* inv_std_dev,
    const T* input,
    int n1,
    int n2,
    double epsilon,
    const T* gamma,
    const T* beta) {
  RowwiseMomentsCUDAKernel<T><<<n1, kCUDABlockReduceNumThreads, 0, stream>>>(
          n2, epsilon, input, mean, inv_std_dev);

  LayerNormForwardCUDAKernel<T><<<n1, kCUDANumThreads, 0, stream>>>(
      n2, input, mean, inv_std_dev, gamma, beta, output);
}

#define LAYERNORM_LINEAR_IMPL(T) \
  template void LaunchLayerNorm<T>(const hipDeviceProp_t& prop, hipStream_t stream, T* output, T* mean, T* inv_std_dev, const T* input, \
                                   int n1, int n2, double epsilon, const T* gamma, const T* beta);

LAYERNORM_LINEAR_IMPL(float)

/*
template <typename T>
void LayerNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    T eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  DCHECK_EQ(X.numel(), M * N);
  DCHECK(!gamma.defined() || gamma.numel() == N);
  DCHECK(!beta.defined() || beta.numel() == N);
  const T* X_data = X.data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.data_ptr<T>() : nullptr;
  const T* beta_data = beta.defined() ? beta.data_ptr<T>() : nullptr;
  T* Y_data = Y->data_ptr<T>();
  T* mean_data = mean->data_ptr<T>();
  T* rstd_data = rstd->data_ptr<T>();

  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();

  RowwiseMomentsCUDAKernel<T>
      <<<M, kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          N, eps, X_data, mean_data, rstd_data);

  C10_CUDA_KERNEL_LAUNCH_CHECK();

  LayerNormForwardCUDAKernel<T><<<M, kCUDANumThreads, 0, hip_stream>>>(
      N, X_data, mean_data, rstd_data, gamma_data, beta_data, Y_data);

  C10_CUDA_KERNEL_LAUNCH_CHECK();
}



void LayerNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    double eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "LayerNormKernelImpl",
      [&]() {
        LayerNormKernelImplInternal<scalar_t>(
            X, gamma, beta, M, N, static_cast<scalar_t>(eps), Y, mean, rstd);
      });
}

} // namespace
*/

/*
std::tuple<Tensor, Tensor, Tensor> layer_norm_cuda(
    const Tensor& input,
    IntArrayRef normalized_shape,
    const c10::optional<Tensor>& weight_opt // optional,
    const c10::optional<Tensor>& bias_opt  // optional,
    double eps) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;
  c10::MaybeOwned<Tensor> bias_maybe_owned =
      at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  auto M_N = _check_layer_norm_inputs(input, normalized_shape, weight, bias);
  auto M = M_N.first;
  auto N = M_N.second;
  auto X = input.expect_contiguous();
  auto gamma = weight.expect_contiguous();
  auto beta = bias.expect_contiguous();

  Tensor Y = at::native::empty_like(
      *X,
      c10::nullopt, // dtype,
      c10::nullopt, // layout
      c10::nullopt, // device
      c10::nullopt, // pin_memory
      LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor mean = at::empty({M}, X->options());
  Tensor rstd = at::empty({M}, X->options());
  if (M > 0) {
    LayerNormKernelImpl(*X, *gamma, *beta, M, N, eps, &Y, &mean, &rstd);

    const auto input_shape = input.sizes();
    const size_t axis = input.dim() - normalized_shape.size();

    std::vector<int64_t> stat_shape;
    for (size_t idx = 0; idx < axis; ++idx) {
      stat_shape.push_back(input_shape[idx]);
    }
    for (size_t idx = axis; idx < input.dim(); ++idx) {
      stat_shape.push_back(1);
    }

    mean = mean.view(stat_shape);
    rstd = rstd.view(stat_shape);
  }
  return std::make_tuple(std::move(Y), std::move(mean), std::move(rstd));
}

REGISTER_DISPATCH(LayerNormKernel, &LayerNormKernelImpl);
*/

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
