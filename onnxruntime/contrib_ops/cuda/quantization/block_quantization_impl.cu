#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "block_quantization_impl.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_utils.h"

#include <hipcub/hipcub.hpp>

namespace onnxruntime {
namespace contrib {
namespace cuda {

typedef struct __align__(8) {
   half x;
   half y;
   half z;
   half w;
} half4;

__device__ __forceinline__ float warpReduceMaxMultipleGroup(float max_value, const int width) {
    constexpr unsigned FULLMASK = 0xFFFFFFFF;
    switch (width) {
        case 32: max_value = max(max_value, __shfl_xor_sync(FULLMASK, max_value, 16));
        case 16: max_value = max(max_value, __shfl_xor_sync(FULLMASK, max_value, 8));
        case 8: max_value = max(max_value, __shfl_xor_sync(FULLMASK, max_value, 4));
        case 4: max_value = max(max_value, __shfl_xor_sync(FULLMASK, max_value, 2));
        case 2: max_value = max(max_value, __shfl_xor_sync(FULLMASK, max_value, 1));
        default: break;
    }
    return max_value;
}

static constexpr int INT8MAX = 127;

// vectorized read/write value in 4 each thread.
// when datablock size <= 128, each warp will handle one full datablock or more than one full datablocks.
template<typename T, typename TInVec4>
__global__ void
BlockQuantizeKernelInWarp(const T* x, unsigned datablock_size, unsigned datablock_count, T* scale, int8_t* y) {
    const int64_t offset = ((int64_t)blockIdx.x * blockDim.x + threadIdx.x) * 4;
    const int64_t datablock_id = offset / datablock_size;
    if (datablock_id >= datablock_count) return;

    TInVec4 x4 = *(const TInVec4*)(x + offset);
    float max_abs_val = fabsf((float)x4.x);
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.y));
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.z));
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.w));

    const int width = static_cast<int>(datablock_size >> 2);
    max_abs_val = warpReduceMaxMultipleGroup(max_abs_val, width);
    const float block_scale_value = max_abs_val / INT8MAX;

    char4 y4{0, 0, 0, 0};
    if (block_scale_value) {
        y4.x = static_cast<char>(__float2int_rn((float)x4.x / block_scale_value));
        y4.y = static_cast<char>(__float2int_rn((float)x4.y / block_scale_value));
        y4.z = static_cast<char>(__float2int_rn((float)x4.z / block_scale_value));
        y4.w = static_cast<char>(__float2int_rn((float)x4.w / block_scale_value));
    }
    *(char4 *)(y + offset) = y4;

    if (offset == datablock_id * datablock_size) { // first thread for a datablock
        scale[datablock_id] = (T)block_scale_value;
    }
}

// vectorized read/write value in 4 each thread.
// when datablock size >= 256, one cuda block will handle on datablock.
template<typename T, typename TInVec4, int TPB>
__global__ void
BlockQuantizeKernelCrossWarp(const T* x, unsigned datablock_size, T* scale, int8_t* y) {
    if (threadIdx.x * 4 >= datablock_size) return;
    const int64_t datablock_id = blockIdx.x;
    const int64_t offset = (datablock_id * datablock_size + threadIdx.x) * 4;

    __shared__ float block_scale_value;
    TInVec4 x4 = *(const TInVec4*)(x + offset);
    float max_abs_val = 0.0f;
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.x));
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.y));
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.z));
    max_abs_val = fmaxf(max_abs_val, fabsf((float)x4.w));

    typedef hipcub::BlockReduce<float, TPB> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    float block_max_abs_val = BlockReduce(temp_storage).Reduce(max_abs_val, hipcub::Max());
    if (threadIdx.x == 0) {
        block_scale_value = block_max_abs_val / INT8MAX;
    }
    __syncthreads();

    char4 y4{0, 0, 0, 0};
    if (scale) {
        y4.x = static_cast<char>(__float2int_rn((float)x4.x / block_scale_value));
        y4.y = static_cast<char>(__float2int_rn((float)x4.y / block_scale_value));
        y4.z = static_cast<char>(__float2int_rn((float)x4.z / block_scale_value));
        y4.w = static_cast<char>(__float2int_rn((float)x4.w / block_scale_value));
    }
    *(char4 *)(y + offset) = y4;

    if (offset == datablock_id * datablock_size) { // first thread for a datablock
        scale[datablock_id] = (T)block_scale_value;
    }
}

template <>
Status CudaBlockQuantize(
    hipStream_t stream,
    const hipDeviceProp_t& /*device_prop*/,
    const half* x,
    unsigned const datablock_size,
    unsigned const datablock_count,
    half* scale,
    int8_t* y) {
  if (datablock_size <= 128) {
    constexpr unsigned TPB = 256;
    constexpr unsigned EPB = TPB * 4;
    const unsigned cuda_blocks = ((int64_t)(datablock_size) * datablock_count + EPB - 1) / EPB;
    BlockQuantizeKernelInWarp<half, half4><<<cuda_blocks, TPB, 0, stream>>>(
        x, datablock_size, datablock_count, scale, y
    );
  } else if (datablock_size == 256) {
    constexpr unsigned TPB = 256 / 4;
    BlockQuantizeKernelCrossWarp<half, half4, TPB><<<datablock_count, TPB, 0, stream>>>(
        x, datablock_size, scale, y
    );
  } else if (datablock_size == 512) {
    constexpr unsigned TPB = 512 / 4;
    BlockQuantizeKernelCrossWarp<half, half4, TPB><<<datablock_count, TPB, 0, stream>>>(
        x, datablock_size, scale, y
    );
  } else if (datablock_size == 1024) {
    constexpr unsigned TPB = 1024 / 4;
    BlockQuantizeKernelCrossWarp<half, half4, TPB><<<datablock_count, TPB, 0, stream>>>(
        x, datablock_size, scale, y
    );
  } else {
    ORT_ENFORCE(false, "Datablock size too large, currently not supported!");
  }

  return CUDA_CALL(hipGetLastError());
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
