#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// The ROCM kernel is hipified from CUDA kernel.
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/rocm_common.h"
#include "contrib_ops/rocm/diffusion/group_norm_common.h"
#include "contrib_ops/rocm/diffusion/group_norm_impl.h"
#include "contrib_ops/rocm/transformers/dump_rocm_tensor.h"

namespace onnxruntime {
namespace contrib {
namespace rocm {

static inline int32_t divUp(int32_t m, int32_t n) {
  return (m + n - 1) / n;
}

static inline __device__ __host__ float sigmoid(float x) {
  return 1.F / (1.F + expf(-x));
}

struct GroupSums {
  // Is it the 1st element of the group?
  int32_t flag;
  // The sum.
  float sum;
  // The sum of squares.
  float sumSq;
};

struct GroupSumsOp {
  inline __device__ GroupSums operator()(GroupSums const& a, GroupSums const& b) {
    GroupSums dst;
    dst.sum = b.flag ? b.sum : (a.sum + b.sum);
    dst.sumSq = b.flag ? b.sumSq : (a.sumSq + b.sumSq);
    dst.flag = a.flag + b.flag;
    return dst;
  }
};

template <typename T, typename U, int32_t ILP>
inline __device__ void UpdateSum(const T* src, int64_t offset, U& sum, U& sumSq) {
  using VecT = onnxruntime::rocm::aligned_vector<T, ILP>;
  const VecT input_v = *reinterpret_cast<const VecT*>(src + offset);

#pragma unroll
  for (int i = 0; i < ILP; i++) {
    const U val = static_cast<U>(input_v.val[i]);
    sum += val;
    sumSq += val * val;
  }
}

template <typename T, int32_t tTHREADS_PER_BLOCK, int32_t ILP>
__global__ void groupNormNHWCSumKernel(const T* src, float* redBuffer, int32_t cPerBlock, int32_t hwPerBlock, int32_t hw,
                                       int32_t hwc, int32_t c, int32_t cPerGroup, int32_t groups, int32_t groupsPerBlock) {
  // The object in charge of doing the sums for the different blocks.
  typedef hipcub::BlockScan<GroupSums, tTHREADS_PER_BLOCK> BlockScan;

  // Allocate shared memory for BlockScan.
  __shared__ typename BlockScan::TempStorage tempStorage;
  // Allocate shared memory for the groups. We could reduce the amount of shared
  // memory reserved.
  __shared__ float2 smem[tTHREADS_PER_BLOCK];

  // The instance in the batch.
  int32_t ni = blockIdx.z;
  // The channel loaded by that thread (ILP channels per thread).
  int32_t ci = blockIdx.x * cPerBlock + threadIdx.x * ILP;

  // The first activation loaded by that block.
  int32_t hwBegin = blockIdx.y * hwPerBlock;
  // The last activation loaded by that block.
  int32_t hwEnd = min(hwBegin + hwPerBlock, hw);

  // The sums.
  float sum = 0.F;
  float sumSq = 0.F;

  // Iterate over the activations to compute the sums.
  if (ci < c) {
    for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi) {
      // The offset.
      int64_t offset = static_cast<int64_t>(ni) * hwc + static_cast<int64_t>(hwi) * c + ci;
      UpdateSum<T, float, ILP>(src, offset, sum, sumSq);
    }
  }

  // The group that thread works on and the channel in the group (modulus).
  int32_t gi = threadIdx.x * ILP / cPerGroup;
  int32_t cj = threadIdx.x * ILP - cPerGroup * gi;

  // The data for the summations.
  GroupSums inp{cj == 0 ? 1 : 0, sum, sumSq};

  // Do the segmented scan.
  GroupSums out;
  BlockScan(tempStorage).InclusiveScan(inp, out, GroupSumsOp());

  // Store the results for the groups in shared memory (to produce coalesced
  // stores later).
  if (cj == cPerGroup - ILP) {  // ILP channels per thread
    smem[gi] = make_float2(out.sum, out.sumSq);
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The global group index.
  int32_t gj = blockIdx.x * groupsPerBlock + threadIdx.x;

  // Threads that have nothing left to do, exit.
  if (threadIdx.x >= groupsPerBlock || gj >= groups) {
    return;
  }

  // The first threads (those storing to global memory, load the values).
  float2 sums = smem[threadIdx.x];

  // Store to global memory.
  atomicAdd(&redBuffer[(2 * ni + 0) * groups + gj], sums.x);
  atomicAdd(&redBuffer[(2 * ni + 1) * groups + gj], sums.y);
}

template <typename T>
void groupNormNHWCSum(const GroupNormNHWCParams<T>* params) {
  // Make sure the values are as we expect.
  ORT_ENFORCE(params->c % params->cPerBlock == 0 && params->hw % params->hwPerBlock == 0);
  // Make sure a group does not span multiple blocks.
  ORT_ENFORCE(params->cPerBlock % params->cPerGroup == 0);

  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params->c / params->cPerBlock;
  // The number of blocks to compute all the activations in a given instance.
  grid.y = divUp(params->hw, params->hwPerBlock);
  // The number of instances.
  grid.z = params->n;

  switch (params->cPerBlock) {
    case 320:
      groupNormNHWCSumKernel<T, 256, 2><<<grid, 256, 0, params->stream>>>(params->src, params->redBuffer, params->cPerBlock, params->hwPerBlock,
                                                                          params->hw, params->hwc, params->c, params->cPerGroup, params->groups, params->groupsPerBlock);
      break;
    case 480:
      groupNormNHWCSumKernel<T, 256, 2><<<grid, 256, 0, params->stream>>>(params->src, params->redBuffer, params->cPerBlock, params->hwPerBlock,
                                                                          params->hw, params->hwc, params->c, params->cPerGroup, params->groups, params->groupsPerBlock);
      break;
    case 256:
      groupNormNHWCSumKernel<T, 128, 2><<<grid, 128, 0, params->stream>>>(params->src, params->redBuffer, params->cPerBlock, params->hwPerBlock,
                                                                          params->hw, params->hwc, params->c, params->cPerGroup, params->groups, params->groupsPerBlock);
      break;
    case 128:
      groupNormNHWCSumKernel<T, 64, 2><<<grid, 64, 0, params->stream>>>(params->src, params->redBuffer, params->cPerBlock, params->hwPerBlock,
                                                                        params->hw, params->hwc, params->c, params->cPerGroup, params->groups, params->groupsPerBlock);
      break;
    default:
      ORT_NOT_IMPLEMENTED("Not implemented");
  }
}

template <typename T, typename U, int32_t ILP>
__device__ void computeGroupNorm(const T* src, T* dst, int64_t offset, U mean, U invStdDev,
                                 const U* gamma_v, const U* beta_v, bool swish) {
  using VecT = onnxruntime::rocm::aligned_vector<T, ILP>;
  const VecT input_v = *reinterpret_cast<const VecT*>(src + offset);
  VecT output_v;

#pragma unroll
  for (int i = 0; i < ILP; i++) {
    U val = static_cast<U>(input_v.val[i]);
    val = (val - mean) * invStdDev;
    val = gamma_v[i] * val + beta_v[i];

    if (swish) {
      val = val * sigmoid(val);
    }
    output_v.val[i] = static_cast<T>(val);
  }
  *(reinterpret_cast<VecT*>(dst + offset)) = output_v;
}

template <typename T, int32_t tTHREADS_PER_BLOCK, int32_t ILP>
__global__ void groupNormNHWCScaleKernel(T* dst, const T* src, const float* gamma, const float* beta, const float* redBuffer, int32_t c, int32_t cPerBlock,
                                         int32_t cPerGroup, int32_t groups, int32_t hwc, float invHWC, int32_t hw, int32_t hwPerBlock, bool withSwish) {
  // The channel loaded by that thread (ILP channels per thread for F16x2).
  int32_t ci = blockIdx.x * cPerBlock + threadIdx.x * ILP;
  if (ci >= c) {
    return;
  }

  // The instance in the batch.
  int32_t ni = blockIdx.z;

  // The group that thread works on and the channel in the group (modulus).
  int32_t gi = ci / cPerGroup;

  // Load the sum and sum of squares for the group.
  float sum = 0.F, sumSq = 0.F;
  if (gi < groups) {
    sum = redBuffer[(2 * ni + 0) * groups + gi];
    sumSq = redBuffer[(2 * ni + 1) * groups + gi];
  }

  using VecF = onnxruntime::rocm::aligned_vector<float, ILP>;

  const VecF gamma_v = *reinterpret_cast<const VecF*>(gamma + ci);
  const VecF beta_v = *reinterpret_cast<const VecF*>(beta + ci);

  // Compute the mean.
  float mean = sum * invHWC;
  // Compute the variance.
  float var = sumSq * invHWC - (mean * mean);
  // Compute the inverse of the stddev.
  float invStdDev = var <= 0.F ? 1.F : rsqrtf(var);

  // The first activation loaded by that block.
  int32_t hwBegin = blockIdx.y * hwPerBlock;
  // The last activation loaded by that block.
  int32_t hwEnd = min(hwBegin + hwPerBlock, hw);

  // Iterate over the activations to compute the sums.
  for (int32_t hwi = hwBegin; hwi < hwEnd; ++hwi) {
    // The src/dst offset.
    int64_t offset = (int64_t)ni * hwc + hwi * c + ci;

    // Fetch ILP channels per thread.
    computeGroupNorm<T, float, ILP>(src, dst, offset, mean, invStdDev, gamma_v.val, beta_v.val, withSwish);
  }
}

template <typename T>
void groupNormNHWCScale(const GroupNormNHWCParams<T>* params) {
  // Make sure the dimensions are aligned with what we expect.
  ORT_ENFORCE(params->c % params->cPerBlock == 0);
  // Make sure a group does not span multiple blocks.
  ORT_ENFORCE(params->cPerBlock % params->cPerGroup == 0);

  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params->c / params->cPerBlock;
  // The number of blocks to compute all the activations in a given instance.
  grid.y = divUp(params->hw, params->hwPerBlock);
  // The number of instances.
  grid.z = params->n;

  switch (params->cPerBlock) {
    case 320:
      groupNormNHWCScaleKernel<T, 256, 2><<<grid, 256, 0, params->stream>>>(params->dst, params->src, params->gamma, params->beta, params->redBuffer, params->c, params->cPerBlock,
                                                                            params->cPerGroup, params->groups, params->hwc, params->invHWC, params->hw, params->hwPerBlock, params->withSwish);
      break;
    case 480:
      groupNormNHWCScaleKernel<T, 256, 2><<<grid, 256, 0, params->stream>>>(params->dst, params->src, params->gamma, params->beta, params->redBuffer, params->c, params->cPerBlock,
                                                                            params->cPerGroup, params->groups, params->hwc, params->invHWC, params->hw, params->hwPerBlock, params->withSwish);
      break;
    case 256:
      groupNormNHWCScaleKernel<T, 128, 2><<<grid, 128, 0, params->stream>>>(params->dst, params->src, params->gamma, params->beta, params->redBuffer, params->c, params->cPerBlock,
                                                                            params->cPerGroup, params->groups, params->hwc, params->invHWC, params->hw, params->hwPerBlock, params->withSwish);
      break;
    case 128:
      groupNormNHWCScaleKernel<T, 64, 2><<<grid, 64, 0, params->stream>>>(params->dst, params->src, params->gamma, params->beta, params->redBuffer, params->c, params->cPerBlock,
                                                                          params->cPerGroup, params->groups, params->hwc, params->invHWC, params->hw, params->hwPerBlock, params->withSwish);
      break;
    default:
      ORT_NOT_IMPLEMENTED("Not implemented");
  }
}

int32_t findMaxDivisor(int32_t n, int32_t maxAllowedDivisor) {
  int32_t maxDivisor = -1;
  for (int32_t i = 1; i <= std::sqrt(n); i++) {
    if (n % i == 0) {
      int32_t divisor1 = n / i;
      int32_t divisor2 = i;

      if (divisor1 > maxDivisor && divisor1 < maxAllowedDivisor) {
        maxDivisor = divisor1;
      }
      if (divisor2 > maxDivisor && divisor2 < maxAllowedDivisor) {
        maxDivisor = divisor2;
      }
    }
  }
  return maxDivisor;
}

template <typename T>
Status LaunchGroupNormKernel(
    hipStream_t stream,
    T* output,
    const T* input,
    const float* gamma,
    const float* beta,
    void* workspace,
    float epsilon,
    int batch_size,
    int num_channels,
    int height,
    int width,
    int num_groups,
    bool use_swish_activation) {
  if (batch_size > static_cast<int>(kMaxGroupNormBatchSize)) {
    return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                           "only support batch_size <= 32. Got", batch_size);
  }

  if (num_groups != static_cast<int>(kGroupNormNumberOfGroups)) {
    return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                           "only num_groups=32 is supported. Got", num_groups);
  }

  int32_t cPerBlock = 320;
  int32_t maxBlocksPerHW = 1024;
  switch (num_channels) {
    case 960:
    case 1920:
      cPerBlock = 480;
      break;
    case 512:
    case 256:
      cPerBlock = 256;
      break;
    case 128:
      cPerBlock = 128;
      break;
    default:
      cPerBlock = 320;
  }

  GroupNormNHWCParams<T> params(nullptr, stream, output, input, gamma, beta, batch_size, height, width, num_channels, num_groups, use_swish_activation);

  params.redBuffer = reinterpret_cast<float*>(workspace);
  params.hw = params.h * params.w;
  const int32_t blocksPerHW = findMaxDivisor(params.hw, maxBlocksPerHW);
  params.hwPerBlock = divUp(params.hw, blocksPerHW);
  params.cPerBlock = cPerBlock;
  params.cPerGroup = params.c / params.groups;
  params.hwc = params.hw * params.c;
  params.invHWC = 1.F / (float)(params.hw * params.cPerGroup);
  params.groupsPerBlock = cPerBlock / params.cPerGroup;

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("input", input, batch_size, num_channels, height * width);
  DUMP_TENSOR("gamma", gamma, 1, num_channels);
  DUMP_TENSOR("beta", beta, 1, num_channels);
  HIP_RETURN_IF_ERROR(hipMemsetAsync(params.redBuffer, 0, GetGroupNormWorkspaceSizeInBytes(), stream));
  groupNormNHWCSum<T>(&params);
  DUMP_TENSOR("workspace", params.redBuffer, batch_size, num_groups, 2);
  HIP_RETURN_IF_ERROR(hipGetLastError());
  groupNormNHWCScale<T>(&params);
  HIP_RETURN_IF_ERROR(hipGetLastError());
  DUMP_TENSOR("output", output, batch_size, num_channels, height * width);
  return Status::OK();
}

template Status LaunchGroupNormKernel<half>(hipStream_t stream, half* output,
                                            const half* input, const float* gamma, const float* beta, void* workspace,
                                            float epsilon, int batch_size, int num_channels,
                                            int height, int width, int num_groups, bool swish);

template Status LaunchGroupNormKernel<float>(hipStream_t stream, float* output,
                                             const float* input, const float* gamma, const float* beta, void* workspace,
                                             float epsilon, int batch_size, int num_channels,
                                             int height, int width, int num_groups, bool swish);
}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
